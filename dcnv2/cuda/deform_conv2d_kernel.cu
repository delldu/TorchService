#include "hip/hip_runtime.h"
/*!
 ******************* BEGIN Caffe Copyright Notice and Disclaimer
 *****************
 *
 * COPYRIGHT
 *
 * All contributions by the University of California:
 * Copyright (c) 2014-2017 The Regents of the University of California (Regents)
 * All rights reserved.
 *
 * All other contributions:
 * Copyright (c) 2014-2017, the respective contributors
 * All rights reserved.
 *
 * Caffe uses a shared copyright model: each contributor holds copyright over
 * their contributions to Caffe. The project versioning records all such
 * contribution and copyright details. If a contributor wants to further mark
 * their specific copyright on a particular contribution, they should indicate
 * their copyright solely in the commit message of the change when it is
 * committed.
 *
 * LICENSE
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 *this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
 *FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 *DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 *SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 *OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 *OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * CONTRIBUTION AGREEMENT
 *
 * By contributing to the BVLC/caffe repository through pull-request, comment,
 * or otherwise, the contributor releases their content to the
 * license and copyright terms herein.
 *
 ***************** END Caffe Copyright Notice and Disclaimer
 *********************
 *
 * Copyright (c) 2018 Microsoft
 * Licensed under The MIT License [see LICENSE for details]
 * \file modulated_deformable_im2col.cuh
 * \brief Function definitions of converting an image to
 * column matrix based on kernel, padding, dilation, and offset.
 * These functions are mainly used in deformable convolution operators.
 * \ref: https://arxiv.org/abs/1703.06211
 * \author Yuwen Xiong, Haozhi Qi, Jifeng Dai, Xizhou Zhu, Han Hu, Dazhi Cheng
 */

// modified from
// https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/blob/mmdetection/mmdet/ops/dcn/src/deform_conv_cuda_kernel.cu

// modified from
// https://github.com/open-mmlab/mmdetection/blob/master/mmdet/ops/dcn/src/deform_conv_cuda.cpp

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
//#include <torch/library.h>
#include <THC/THCAtomics.cuh>

#include "cuda_helpers.h"
#include "deform_conv2d.h"


const int kMaxParallelImgs = 32;

inline unsigned int GET_THREADS() {
#ifdef __HIP_PLATFORM_HCC__
  return 256;
#endif
  if (at::cuda::getCurrentDeviceProperties()->major >= 6) {
    return 1024;
  }
  return 512;
}

inline unsigned int GET_BLOCKS(
    const unsigned int THREADS,
    const unsigned int N) {
  unsigned int kMaxGridNum =
      at::cuda::getCurrentDeviceProperties()->maxGridSize[0];
  return std::min(kMaxGridNum, (N + THREADS - 1) / THREADS);
}

template <typename scalar_t>
__device__ scalar_t bilinear_interpolate(
    const scalar_t* in,
    int height,
    int width,
    scalar_t h,
    scalar_t w) {
  if (h <= -1 || height <= h || w <= -1 || width <= w) {
    return 0;
  }

  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = in[h_low * width + w_low];
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = in[h_low * width + w_high];
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = in[h_high * width + w_low];
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = in[h_high * width + w_high];

  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__global__ void deformable_im2col_cuda_kernel(
    int n,
    const scalar_t* input_ptr,
    const scalar_t* offset_ptr,
    const scalar_t* mask_ptr,
    int height,
    int width,
    int weight_h,
    int weight_w,
    int pad_h,
    int pad_w,
    int stride_h,
    int stride_w,
    int dilation_h,
    int dilation_w,
    int batch_sz,
    int n_in_channels,
    int n_offset_grps,
    int out_h,
    int out_w,
    bool use_mask,
    scalar_t* columns_ptr) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int out_x = index % out_w;
    const int out_y = (index / out_w) % out_h;
    const int out_b = (index / (out_w * out_h)) % batch_sz;
    const int in_c = index / (out_w * out_h * batch_sz);
    const int out_c = in_c * weight_h * weight_w;

    int c_per_offset_grp = n_in_channels / n_offset_grps;
    const int grp_idx = in_c / c_per_offset_grp;

    columns_ptr +=
        (out_c * (batch_sz * out_h * out_w) + out_b * (out_h * out_w) +
         out_y * out_w + out_x);

    input_ptr +=
        (out_b * (n_in_channels * height * width) + in_c * (height * width));

    offset_ptr += (out_b * n_offset_grps + grp_idx) * 2 * weight_h * weight_w *
        out_h * out_w;

    if (use_mask) {
      mask_ptr += (out_b * n_offset_grps + grp_idx) * weight_h * weight_w *
          out_h * out_w;
    }

    for (int i = 0; i < weight_h; ++i) {
      for (int j = 0; j < weight_w; ++j) {
        const int mask_idx = i * weight_w + j;
        const int offset_idx = 2 * mask_idx;

        scalar_t mask_value = 1;
        if (use_mask) {
          mask_value =
              mask_ptr[mask_idx * (out_h * out_w) + out_y * out_w + out_x];
        }

        const scalar_t offset_h =
            offset_ptr[offset_idx * (out_h * out_w) + out_y * out_w + out_x];
        const scalar_t offset_w = offset_ptr
            [(offset_idx + 1) * (out_h * out_w) + out_y * out_w + out_x];
        const scalar_t y =
            (out_y * stride_h - pad_h) + i * dilation_h + offset_h;
        const scalar_t x =
            (out_x * stride_w - pad_w) + j * dilation_w + offset_w;
        *columns_ptr =
            mask_value * bilinear_interpolate(input_ptr, height, width, y, x);
        columns_ptr += batch_sz * out_h * out_w;
      }
    }
  }
}

void deformable_im2col_cuda_app(
    const at::Tensor& input,
    const at::Tensor& data_offset,
    const at::Tensor& data_mask,
    int n_in_channels,
    int height,
    int width,
    int weight_h,
    int weight_w,
    int pad_h,
    int pad_w,
    int stride_h,
    int stride_w,
    int dilation_h,
    int dilation_w,
    int out_h,
    int out_w,
    int parallel_imgs,
    int deformable_group,
    bool use_mask,
    at::Tensor data_col) {
  int num_kernels = n_in_channels * out_h * out_w * parallel_imgs;

  const unsigned int threads = GET_THREADS();
  const unsigned int blocks = GET_BLOCKS(threads, num_kernels);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "deformable_im2col_cuda_app", ([&] {
        deformable_im2col_cuda_kernel<<<blocks, threads>>>(
            num_kernels,
            input.data_ptr<scalar_t>(),
            data_offset.data_ptr<scalar_t>(),
            data_mask.data_ptr<scalar_t>(),
            height,
            width,
            weight_h,
            weight_w,
            pad_h,
            pad_w,
            stride_h,
            stride_w,
            dilation_h,
            dilation_w,
            parallel_imgs,
            n_in_channels,
            deformable_group,
            out_h,
            out_w,
            use_mask,
            data_col.data_ptr<scalar_t>());
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_im2col_cuda_app: %s\n", hipGetErrorString(err));
  }
}

static int get_greatest_divisor_below_bound(int n, int bound) {
  for (int k = bound; k > 1; --k) {
    if (n % k == 0) {
      return k;
    }
  }
  return 1;
}

at::Tensor deform_conv2d_cuda_forward_kernel(
    const at::Tensor& input,
    const at::Tensor& weight,
    const at::Tensor& offset,
    const at::Tensor& mask,
    const at::Tensor& bias,
    int64_t stride_h,
    int64_t stride_w,
    int64_t pad_h,
    int64_t pad_w,
    int64_t dilation_h,
    int64_t dilation_w,
    int64_t n_weight_grps,
    int64_t n_offset_grps,
    bool use_mask) {
  at::Tensor input_c = input.contiguous();
  at::Tensor offset_c = offset.contiguous();
  at::Tensor weight_c = weight.contiguous();
  at::Tensor mask_c = mask.contiguous();
  at::Tensor bias_c = bias.contiguous();

  TORCH_CHECK(input_c.ndimension() == 4);
  TORCH_CHECK(offset_c.ndimension() == 4);
  TORCH_CHECK(!use_mask || mask_c.ndimension() == 4);
  TORCH_CHECK(weight_c.ndimension() == 4);
  TORCH_CHECK(input_c.is_cuda(), "input must be a CUDA tensor");

  at::DeviceGuard guard(input_c.device());

  int batch_sz = input_c.size(0);
  int in_channels = input_c.size(1);
  int in_h = input_c.size(2);
  int in_w = input_c.size(3);

  int n_parallel_imgs =
      get_greatest_divisor_below_bound(batch_sz, kMaxParallelImgs);

  int out_channels = weight_c.size(0);
  int weight_h = weight_c.size(2);
  int weight_w = weight_c.size(3);

  int ker_h = dilation_h * (weight_h - 1) + 1;
  int ker_w = dilation_w * (weight_w - 1) + 1;
  int out_h = ((in_h + 2 * pad_h - ker_h) / stride_h) + 1;
  int out_w = ((in_w + 2 * pad_w - ker_w) / stride_w) + 1;

  TORCH_CHECK(
      weight_h > 0 && weight_w > 0,
      "weight_h: ",
      weight_h,
      " weight_w: ",
      weight_w);
  TORCH_CHECK(
      stride_h > 0 && stride_w > 0,
      "stride_h: ",
      stride_h,
      " stride_w: ",
      stride_w);
  TORCH_CHECK(pad_h >= 0 && pad_w >= 0, "pad_h: ", pad_h, " pad_w: ", pad_w);
  TORCH_CHECK(
      dilation_h > 0 && dilation_w > 0,
      "dilation_h: ",
      dilation_h,
      " dilation_w: ",
      dilation_w);

  TORCH_CHECK(weight_c.size(1) * n_weight_grps == input_c.size(1));
  TORCH_CHECK(weight_c.size(0) % n_weight_grps == 0);
  TORCH_CHECK(
      (offset_c.size(1) == n_offset_grps * 2 * weight_h * weight_w),
      "offset.shape[1] is not valid: got: ",
      offset_c.size(1),
      " expected: ",
      n_offset_grps * 2 * weight_h * weight_w);
  TORCH_CHECK(
      (!use_mask || mask_c.size(1) == n_offset_grps * weight_h * weight_w),
      "mask.shape[1] is not valid: got: ",
      mask_c.size(1),
      " expected: ",
      n_offset_grps * weight_h * weight_w);
  TORCH_CHECK(input_c.size(1) % n_offset_grps == 0);

  TORCH_CHECK(
      (offset_c.size(0) == input_c.size(0)), "invalid batch size of offset");
  TORCH_CHECK(
      (offset_c.size(2) == out_h && offset_c.size(3) == out_w),
      "offset output dims: (",
      offset_c.size(2),
      ", ",
      offset_c.size(3),
      ") - ",
      "computed output dims: (",
      out_h,
      ", ",
      out_w,
      ")");
  TORCH_CHECK(
      (mask_c.size(0) == input_c.size(0)), "invalid batch size of mask");
  TORCH_CHECK(
      (!use_mask || (mask_c.size(2) == out_h && mask_c.size(3) == out_w)),
      "mask output dims: (",
      mask_c.size(2),
      ", ",
      mask_c.size(3),
      ") - ",
      "computed output dims: (",
      out_h,
      ", ",
      out_w,
      ")");
  TORCH_CHECK(
      out_h > 0 && out_w > 0,
      "Calculated output size too small - out_h: ",
      out_h,
      " out_w: ",
      out_w);

  auto out =
      at::zeros({batch_sz, out_channels, out_h, out_w}, input_c.options());
  if (batch_sz == 0) {
    return out;
  }

  // Separate batches into blocks
  out = out.view(
      {batch_sz / n_parallel_imgs,
       n_parallel_imgs,
       out_channels,
       out_h,
       out_w});
  input_c = input_c.view(
      {batch_sz / n_parallel_imgs, n_parallel_imgs, in_channels, in_h, in_w});

  offset_c = offset_c.view(
      {batch_sz / n_parallel_imgs,
       n_parallel_imgs,
       n_offset_grps * 2 * weight_h * weight_w,
       out_h,
       out_w});

  if (use_mask) {
    mask_c = mask_c.view(
        {batch_sz / n_parallel_imgs,
         n_parallel_imgs,
         n_offset_grps * weight_h * weight_w,
         out_h,
         out_w});
  }

  at::Tensor out_buf = at::zeros(
      {batch_sz / n_parallel_imgs,
       out_channels,
       n_parallel_imgs * out_h,
       out_w},
      out.options());

  // Separate channels into convolution groups
  out_buf = out_buf.view(
      {out_buf.size(0),
       n_weight_grps,
       out_buf.size(1) / n_weight_grps,
       out_buf.size(2),
       out_buf.size(3)});
  weight_c = weight_c.view(
      {n_weight_grps,
       weight_c.size(0) / n_weight_grps,
       weight_c.size(1),
       weight_c.size(2),
       weight_c.size(3)});

  // Sample points and perform convolution
  auto columns = at::zeros(
      {in_channels * weight_h * weight_w, n_parallel_imgs * out_h * out_w},
      input_c.options());
  for (int b = 0; b < batch_sz / n_parallel_imgs; b++) {
    deformable_im2col_cuda_app(
        input_c[b],
        offset_c[b],
        mask_c[b],
        in_channels,
        in_h,
        in_w,
        weight_h,
        weight_w,
        pad_h,
        pad_w,
        stride_h,
        stride_w,
        dilation_h,
        dilation_w,
        out_h,
        out_w,
        n_parallel_imgs,
        n_offset_grps,
        use_mask,
        columns);

    columns = columns.view(
        {n_weight_grps, columns.size(0) / n_weight_grps, columns.size(1)});
    for (int g = 0; g < n_weight_grps; g++) {
      out_buf[b][g] = out_buf[b][g]
                          .flatten(1)
                          .addmm_(weight_c[g].flatten(1), columns[g])
                          .view_as(out_buf[b][g]);
    }
    columns =
        columns.view({columns.size(0) * columns.size(1), columns.size(2)});
  }

  out_buf = out_buf.view(
      {batch_sz / n_parallel_imgs,
       out_channels,
       n_parallel_imgs,
       out_h,
       out_w});
  out_buf.transpose_(1, 2);
  out.copy_(out_buf);
  out = out.view({batch_sz, out_channels, out_h, out_w});

  return out + bias_c.view({1, out_channels, 1, 1});
}
